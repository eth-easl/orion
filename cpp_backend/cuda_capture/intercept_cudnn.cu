
#include "intercept_temp.h"

void getDescriptor(const hipdnnTensorDescriptor_t desc) {

	int ndims = 10;
	hipdnnDataType_t* dtype = (hipdnnDataType_t*)malloc(sizeof(hipdnnDataType_t));
	int* nbdims = (int*)malloc(sizeof(int));
	int dimA[10] = {0};
	int strideA[10] = {0};

	hipdnnStatus_t status = hipdnnGetTensorNdDescriptor(desc, ndims, dtype, nbdims, dimA, strideA);
	printf("%d\n", *dtype);

	assert (status==HIPDNN_STATUS_SUCCESS);

}

hipdnnStatus_t hipdnnConvolutionForward(hipdnnHandle_t handle, const void *alpha, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionFwdAlgo_t algo, void *workSpace, size_t workSpaceSizeInBytes, const void *beta, const hipdnnTensorDescriptor_t yDesc, void *y) {


	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	// if (idx < 2)
	// 	block(idx,  mutexes, kqueues);

	cudnnConvolutionForward_record new_conv_record = {
		handle,
		alpha,
		xDesc,
		x,
		wDesc,
		w,
		convDesc,
		algo,
		workSpace,
		workSpaceSizeInBytes,
		beta,
		yDesc,
		y
	};
	union func_data new_func_data;
	new_func_data.cudnnConvRecord = new_conv_record;
	func_record new_record = {CUDNN_CONV_RECORD, new_func_data};

	// push or run
	if (idx < 2) {
		 pthread_mutex_lock(mutexes[idx]);

		 DEBUG_PRINT("[INTERCEPTER-CATCH-%d]-[%d] Caught hipdnnConvolutionForward, CUDNN handle is %p\n", idx, func_indexes[idx], handle, idx);

		 kqueues[idx]->push(new_record);
		 func_indexes[idx] += 1;
		 pthread_mutex_unlock(mutexes[idx]);

		 block(idx, mutexes, kqueues);
	}
	else {

		if (cudnn_conv_func==NULL) {
			*(void **)(&cudnn_conv_func) = dlsym(RTLD_NEXT, "hipdnnConvolutionForward");
			assert(cudnn_conv_func != NULL);
		}

		status = (*cudnn_conv_func)(handle, alpha, xDesc, x, wDesc, w, convDesc, algo, workSpace, workSpaceSizeInBytes, beta, yDesc, y);
		if (status != HIPDNN_STATUS_SUCCESS)
			printf("status is %d\n", status);
		assert (status == HIPDNN_STATUS_SUCCESS);

		DEBUG_PRINT("CONV submitted!!\n");
	}

	return status;

}

hipdnnStatus_t cudnnBatchNormalizationForwardTrainingEx(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, cudnnBatchNormOps_t bnOps, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *xData, const hipdnnTensorDescriptor_t zDesc,  const void *zData, const hipdnnTensorDescriptor_t yDesc, void *yData, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScaleData, const void *bnBiasData, double exponentialAverageFactor, void *resultRunningMeanData, void *resultRunningVarianceData, double epsilon, void *saveMean, void *saveInvVariance, const hipdnnActivationDescriptor_t activationDesc,  void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes) {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	// if (idx < 2)
	// 	block(idx,  mutexes, kqueues);

	// create record
	cudnnBatchNormalizationForwardTrainingEx_record new_bn_record = {
		handle,
		mode,
		bnOps,
		alpha,
		beta,
		xDesc,
		xData,
		zDesc,
		zData,
		yDesc,
		yData,
		bnScaleBiasMeanVarDesc,
		bnScaleData,
		bnBiasData,
		exponentialAverageFactor,
		resultRunningMeanData,
		resultRunningVarianceData,
		epsilon,
		saveMean,
		saveInvVariance,
		activationDesc,
		workspace,
		workSpaceSizeInBytes,
		reserveSpace,
		reserveSpaceSizeInBytes

	};
	union func_data new_func_data;
	new_func_data.cudnnBNormRecord = new_bn_record;
	func_record new_record = {CUDNN_BNORM_RECORD, new_func_data};

	// push or run

	if (idx < 2) {

		pthread_mutex_lock(mutexes[idx]);
		DEBUG_PRINT("[INTERCEPTER-CATCH-%d]-[%d] Caught cudnnBatchNormalizationForwardTrainingEx, handle is %p\n", idx, func_indexes[idx], handle);
		kqueues[idx]->push(new_record);
		func_indexes[idx] += 1;

		pthread_mutex_unlock(mutexes[idx]);

		block(idx,  mutexes, kqueues);
	}
	else {

		if (cudnn_bnorm_func==NULL) {
			*(void **)(&cudnn_bnorm_func) = dlsym(RTLD_NEXT, "cudnnBatchNormalizationForwardTrainingEx");
			assert(cudnn_bnorm_func != NULL);
		}
		//printf("run func %p\n", cudnn_bnorm_function);
		status = (*cudnn_bnorm_func)(handle, mode, bnOps, alpha, beta, xDesc, xData, zDesc, zData, yDesc, yData, bnScaleBiasMeanVarDesc, bnScaleData, bnBiasData, exponentialAverageFactor, resultRunningMeanData, resultRunningVarianceData, epsilon, saveMean, saveInvVariance, activationDesc, workspace, workSpaceSizeInBytes, reserveSpace, reserveSpaceSizeInBytes);
		// if (status != HIPDNN_STATUS_SUCCESS)
		// 	printf("status is %d\n", status);
		// assert (status == HIPDNN_STATUS_SUCCESS);

		// DEBUG_PRINT("BNORM submitted!!\n");

	}

	return status;
}


hipdnnStatus_t hipdnnBatchNormalizationForwardInference(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnTensorDescriptor_t yDesc, void *y, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScale, const void *bnBias, const void *estimatedMean, const void *estimatedVariance, double epsilon)

{

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;
	// if (idx < 2)
	// 	block(idx,  mutexes, kqueues);

	// create record
	cudnnBatchNormalizationForwardInference_record bn_record = {
		handle,
		mode,
		alpha,
		beta,
		xDesc,
		x,
		yDesc,
		y,
		bnScaleBiasMeanVarDesc,
		bnScale,
		bnBias,
		estimatedMean,
		estimatedVariance,
		epsilon
	};

	union func_data new_func_data;
	new_func_data.cudnnBNormInfRecord = bn_record;
	func_record new_record = {CUDNN_BNORM_INF_RECORD, new_func_data};

	if (idx < 2) {

		pthread_mutex_lock(mutexes[idx]);
		DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnBatchNormalizationForwardInference, handle is %p, index is %d\n", func_indexes[idx], handle, idx);
		kqueues[idx]->push(new_record);
		func_indexes[idx] += 1;

		pthread_mutex_unlock(mutexes[idx]);

		block(idx,  mutexes, kqueues);

	}
	else {

		if (cudnn_bnorm_infer_func==NULL) {
			*(void **)(&cudnn_bnorm_infer_func) = dlsym(RTLD_NEXT, "hipdnnBatchNormalizationForwardInference");
			assert(cudnn_bnorm_infer_func != NULL);
		}

		status = (*cudnn_bnorm_infer_func)(handle, mode, alpha, beta, xDesc, x, xDesc, y, bnScaleBiasMeanVarDesc, bnScale, bnBias, estimatedMean, estimatedVariance, epsilon);
		assert (status == HIPDNN_STATUS_SUCCESS);

	}

	return status;
}


hipdnnStatus_t hipdnnRNNForwardInference(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes)  {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	if (idx < 2) {

		hipdnnTensorDescriptor_t* xDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
	        //hipdnnStatus_t s = hipdnnCreateTensorDescriptor(xDesc_new);

		*xDesc_new = *xDesc;
		//printf("%p, %p, %p, %p\n", xDesc, *xDesc, xDesc_new, *(xDesc_new));
		//memcpy(xDesc_new, xDesc, sizeof(hipdnnTensorDescriptor_t));

		hipdnnTensorDescriptor_t* yDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
		*yDesc_new = *yDesc;


		cudnnRNNForwardInf_record rnn_record = {
			handle,
			rnnDesc,
			seqLength,
			xDesc_new,
			x,
			hxDesc,
			hx,
			cxDesc,
			cx,
			wDesc,
			w,
			yDesc_new,
			y,
			hyDesc,
			hy,
			cyDesc,
			cy,
			workspace,
			workSpaceSizeInBytes
		};

		union func_data new_func_data;
		new_func_data.cudnnRnnInfRecord = rnn_record;
		func_record new_record = {CUDNN_RNN_INF_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnRNNForwardInference, handle is %p, index is %d\n", func_indexes[idx], handle, idx);
		kqueues[idx]->push(new_record);
		func_indexes[idx] += 1;
		pthread_mutex_unlock(mutexes[idx]);

		block(idx,  mutexes, kqueues);
	}
	else {

		if (cudnn_rnn_func==NULL) {
			*(void **)(&cudnn_rnn_func) = dlsym(RTLD_NEXT, "hipdnnRNNForwardInference");
			assert(cudnn_rnn_func != NULL);
		}

		status = (*cudnn_rnn_func)(handle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSpaceSizeInBytes);

		printf("------------------------- cudnn status is %d\n", status);
		// TODO: not sure why this complains here in just one call!
		assert (status == HIPDNN_STATUS_SUCCESS);

		// hipError_t err_all = hipDeviceSynchronize(); // for debugging
		// CHECK_CUDA_ERROR(err_all);
	}

	return status;

}


hipdnnStatus_t hipdnnRNNForwardTraining(
	hipdnnHandle_t handle,
	const hipdnnRNNDescriptor_t rnnDesc,
	const int seqLength,
	const hipdnnTensorDescriptor_t *xDesc,
	const void *x,
    const hipdnnTensorDescriptor_t hxDesc,
    const void *hx,
    const hipdnnTensorDescriptor_t cxDesc,
    const void *cx,
    const hipdnnFilterDescriptor_t wDesc,
    const void *w,
    const hipdnnTensorDescriptor_t *yDesc,
    void *y,
    const hipdnnTensorDescriptor_t hyDesc,
    void *hy,
    const hipdnnTensorDescriptor_t cyDesc,
    void *cy,
    void *workspace,
    size_t workSpaceSizeInBytes,
    void *reserveSpace,
    size_t reserveSpaceSizeInBytes
) {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	if (idx < 2) {

		hipdnnTensorDescriptor_t* xDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
	        //hipdnnStatus_t s = hipdnnCreateTensorDescriptor(xDesc_new);

		*xDesc_new = *xDesc;
		printf("%p, %p, %p, %p\n", xDesc, *xDesc, xDesc_new, *(xDesc_new));
		//memcpy(xDesc_new, xDesc, sizeof(hipdnnTensorDescriptor_t));

		hipdnnTensorDescriptor_t* yDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
		*yDesc_new = *yDesc;

		cudnnRNNForwardTraining_record rnn_record = {
			handle,
			rnnDesc,
			seqLength,
			xDesc,
			x,
			hxDesc,
			hx,
			cxDesc,
			cx,
			wDesc,
			w,
			yDesc,
			y,
			hyDesc,
			hy,
			cyDesc,
			cy,
			workspace,
			workSpaceSizeInBytes,
			reserveSpace,
			reserveSpaceSizeInBytes
		};

		union func_data new_func_data;
		new_func_data.cudnnRnnTrainRecord = rnn_record;
		func_record new_record = {CUDNN_RNN_TRAIN_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnRNNForwardTraining, handle is %p, index is %d\n", func_indexes[idx], handle, idx);

		kqueues[idx]->push(new_record);
		func_indexes[idx] += 1;
		pthread_mutex_unlock(mutexes[idx]);

		block(idx,  mutexes, kqueues);
	}
	else {

		if (cudnn_rnn_train_func==NULL) {
			*(void **)(&cudnn_rnn_train_func) = dlsym(RTLD_NEXT, "hipdnnRNNForwardTraining");
			assert(cudnn_rnn_train_func != NULL);
		}

		status = (*cudnn_rnn_train_func)(handle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSpaceSizeInBytes, reserveSpace, reserveSpaceSizeInBytes);

		// hipError_t err_all = hipDeviceSynchronize(); // for debugging
		// CHECK_CUDA_ERROR(err_all);
	}

	return status;
}


// backward

hipdnnStatus_t cudnnBatchNormalizationBackwardEx (
    hipdnnHandle_t handle,
    hipdnnBatchNormMode_t mode,
    cudnnBatchNormOps_t bnOps,
    const void *alphaDataDiff,
    const void *betaDataDiff,
    const void *alphaParamDiff,
    const void *betaParamDiff,
    const hipdnnTensorDescriptor_t xDesc,
    const void *xData,
    const hipdnnTensorDescriptor_t yDesc,
    const void *yData,
    const hipdnnTensorDescriptor_t dyDesc,
    const void *dyData,
    const hipdnnTensorDescriptor_t dzDesc,
    void *dzData,
    const hipdnnTensorDescriptor_t dxDesc,
    void *dxData,
    const hipdnnTensorDescriptor_t dBnScaleBiasDesc,
    const void *bnScaleData,
    const void *bnBiasData,
    void *dBnScaleData,
    void *dBnBiasData,
    double epsilon,
    const void *savedMean,
    const void *savedInvVariance,
    const hipdnnActivationDescriptor_t activationDesc,
    void *workspace,
    size_t workSpaceSizeInBytes,
    void *reserveSpace,
    size_t reserveSpaceSizeInBytes
) {

	int idx = get_idx();
	assert (idx >= 0);

	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	cudnnBatchNormalizationBackwardEx_record record = {
		handle,
		mode,
		bnOps,
		alphaDataDiff,
		betaDataDiff,
		alphaParamDiff,
		betaParamDiff,
		xDesc,
		xData,
		yDesc,
		yData,
		dyDesc,
		dyData,
		dzDesc,
		dzData,
		dxDesc,
		dxData,
		dBnScaleBiasDesc,
		bnScaleData,
		bnBiasData,
		dBnScaleData,
		dBnBiasData,
		epsilon,
		savedMean,
		savedInvVariance,
		activationDesc,
		workspace,
		workSpaceSizeInBytes,
		reserveSpace,
		reserveSpaceSizeInBytes
	};

	union func_data new_func_data;
	new_func_data.cudnnBNormBackRecord = record;
	func_record new_record = {CUDNN_BNORM_BACKWARD_RECORD, new_func_data};

	if (idx < 2) {

		pthread_mutex_lock(mutexes[idx]);
		DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught cudnnBatchNormalizationBackwardEx!Index is %d\n", func_indexes[idx], idx);
		kqueues[idx]->push(new_record);
		func_indexes[idx] += 1;
		pthread_mutex_unlock(mutexes[idx]);

		block(idx, mutexes, kqueues);
	}

	else {

		if (cudnn_bnorm_bw_func==NULL) {
			*(void **)(&cudnn_bnorm_bw_func) = dlsym(RTLD_NEXT, "cudnnBatchNormalizationBackwardEx");
			assert(cudnn_bnorm_bw_func != NULL);
		}


		status = (*cudnn_bnorm_bw_func)(
			handle,
			mode,
			bnOps,
			alphaDataDiff,
			betaDataDiff,
			alphaParamDiff,
			betaParamDiff,
			xDesc,
			xData,
			yDesc,
			yData,
			dyDesc,
			dyData,
			dzDesc,
			dzData,
			dxDesc,
			dxData,
			dBnScaleBiasDesc,
			bnScaleData,
			bnBiasData,
			dBnScaleData,
			dBnBiasData,
			epsilon,
			savedMean,
			savedInvVariance,
			activationDesc,
			workspace,
			workSpaceSizeInBytes,
			reserveSpace,
			reserveSpaceSizeInBytes
		);

		if (status != HIPDNN_STATUS_SUCCESS)
			printf("status is %d\n", status);
		assert (status == HIPDNN_STATUS_SUCCESS);

		DEBUG_PRINT("BNORM BACKWARD submitted!!\n");

	}

	return status;
}

hipdnnStatus_t hipdnnConvolutionBackwardData(
    hipdnnHandle_t handle,
    const void *alpha,
    const hipdnnFilterDescriptor_t wDesc,
    const void *w,
    const hipdnnTensorDescriptor_t dyDesc,
    const void *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    hipdnnConvolutionBwdDataAlgo_t algo,
    void *workSpace,
    size_t workSpaceSizeInBytes,
    const void *beta,
    const hipdnnTensorDescriptor_t dxDesc,
    void *dx
)
{

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	if (idx < 2) {
		cudnnConvolutionBackwardData_record record = {
			handle,
			alpha,
			wDesc,
			w,
			dyDesc,
			dy,
			convDesc,
			algo,
			workSpace,
			workSpaceSizeInBytes,
			beta,
			dxDesc,
			dx
		};

		union func_data new_func_data;
		new_func_data.cudnnConvBackDataRecord = record;
		func_record new_record = {CUDNN_CONV_DATA_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnConvolutionBackwardData!Index is %d\n", func_indexes[idx], idx);

		kqueues[idx]->push(new_record);
		func_indexes[idx] += 1;
		pthread_mutex_unlock(mutexes[idx]);

		block(idx,  mutexes, kqueues);
	}
	else {

		if (cudnn_conv_bw_data_func==NULL) {
			*(void **)(&cudnn_conv_bw_data_func) = dlsym(RTLD_NEXT, "hipdnnConvolutionBackwardData");
			assert(cudnn_conv_bw_data_func != NULL);
		}
		status = (*cudnn_conv_bw_data_func)(
			handle,
			alpha,
			wDesc,
			w,
			dyDesc,
			dy,
			convDesc,
			algo,
			workSpace,
			workSpaceSizeInBytes,
			beta,
			dxDesc,
			dx
		);
		assert (status == HIPDNN_STATUS_SUCCESS);
	}

	return status;
}

hipdnnStatus_t hipdnnConvolutionBackwardFilter(
    hipdnnHandle_t handle,
    const void *alpha,
    const hipdnnTensorDescriptor_t xDesc,
    const void *x,
    const hipdnnTensorDescriptor_t dyDesc,
    const void *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    hipdnnConvolutionBwdFilterAlgo_t algo,
    void *workSpace,
    size_t workSpaceSizeInBytes,
    const void *beta,
    const hipdnnFilterDescriptor_t dwDesc,
    void *dw
) {

	int idx = get_idx();
	assert (idx >= 0);

	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	cudnnConvolutionBackwardFilter_record new_conv_record = {
		handle,
		alpha,
		xDesc,
		x,
		dyDesc,
		dy,
		convDesc,
		algo,
		workSpace,
		workSpaceSizeInBytes,
		beta,
		dwDesc,
		dw
	};

	union func_data new_func_data;
	new_func_data.cudnnConvBackFilterRecord = new_conv_record;
	func_record new_record = {CUDNN_CONV_FILTER_RECORD, new_func_data};

	if (idx < 2) {

		pthread_mutex_lock(mutexes[idx]);
		DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnConvolutionBackwardFilter!Index is %d\n", func_indexes[idx], idx);
		kqueues[idx]->push(new_record);
		func_indexes[idx] += 1;
		pthread_mutex_unlock(mutexes[idx]);

		block(idx,  mutexes, kqueues);

	}
	else {

		if (cudnn_conv_bw_filter_func==NULL) {
			*(void **)(&cudnn_conv_bw_filter_func) = dlsym(RTLD_NEXT, "hipdnnConvolutionBackwardFilter");
			assert(cudnn_conv_bw_filter_func != NULL);
		}

		status = (*cudnn_conv_bw_filter_func)(
			handle,
			alpha,
			xDesc,
			x,
			dyDesc,
			dy,
			convDesc,
			algo,
			workSpace,
			workSpaceSizeInBytes,
			beta,
			dwDesc,
			dw
		);

		assert (status == HIPDNN_STATUS_SUCCESS);

	}

	return status;
}


hipdnnStatus_t hipdnnDestroyActivationDescriptor(hipdnnActivationDescriptor_t activationDesc) {
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyRNNDescriptor(hipdnnRNNDescriptor_t rnnDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyRNNDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyTensorDescriptor(hipdnnTensorDescriptor_t tensorDesc) {

	// mock cudnn destroy TensorDescriptor
	//DEBUG_PRINT("Caught a hipdnnDestroyTensorDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}


hipdnnStatus_t hipdnnDestroyFilterDescriptor(hipdnnFilterDescriptor_t filterDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyFilterDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;

}


hipdnnStatus_t hipdnnDestroyConvolutionDescriptor(hipdnnConvolutionDescriptor_t convDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyConvolutionDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyDropoutDescriptor(hipdnnDropoutDescriptor_t dropoutDesc) {
	//DEBUG_PRINT("Caught a hipdnnDestroyDropoutDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;

}


hipdnnStatus_t hipdnnDestroy(hipdnnHandle_t handle) {

	printf("Caught a hipdnnDestroy, Do nothing!\n ");
	return HIPDNN_STATUS_SUCCESS;
}