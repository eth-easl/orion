#include "intercept_temp.h"

using namespace std;
using at::native::ReduceOp;
using at::_isnan;

template <typename acc_t>
struct MaxNanFunctor {
       	__device__ __forceinline__ acc_t operator()(acc_t a, acc_t b) const {
		return (at::_isnan(a) || a > b) ? a : b;
	}
};

template <typename acc_t>
struct MinNanFunctor {
	  __device__ __forceinline__ acc_t operator()(acc_t a, acc_t b) const {
		return (at::_isnan(a) || a < b) ? a : b;
	  }
};

template <typename T>
T* create_new_reduce_arg(void* args0) {

	T* reduce_arg = (T*)args0;
	T* new_reduce_arg = (T*)malloc(sizeof(T));
	char* dst0 = (char*)(reduce_arg->dst[0]);
	char* dst1 = (char*)(reduce_arg->dst[1]);

	*new_reduce_arg = T(
		reduce_arg->ops,
		reduce_arg->config,
		reduce_arg->input_calc,
		reduce_arg->output_calc,
		reduce_arg->src,
		dst0,
		dst1, //check this
		reduce_arg->acc_buf,
		reduce_arg->cta_buf,
		reduce_arg->semaphores,
		reduce_arg->ident,
		reduce_arg->noutputs,
		reduce_arg->base_idx
	);

	return new_reduce_arg;

}


queue<func_record> kqueue0;
queue<func_record> kqueue1;
pthread_mutex_t mutex0;
pthread_mutex_t mutex1;

vector<char*> fnames0;
vector<char*> fnames1;
volatile pid_t thread_ids[5]; // 2*N threads + scheduler

queue<func_record>* kqueues[2] = {&kqueue0, &kqueue1};
pthread_mutex_t* mutexes[2] = {&mutex0, &mutex1};
vector<char*>* func_names[2] = {&fnames0, &fnames1};
char* model_names[2];

int func_indexes[2] = {0, 0};

hipStream_t client_streams[2];
bool streams_set[2] = {false, false};

using namespace boost::interprocess;

// new
volatile int* shmem = NULL;
volatile int* streams_shmem = NULL;
mapped_region* region;
mapped_region* streams_region;
hipStream_t lp_stream;
hipStream_t hp_stream;

hipEvent_t lp_event;
hipEvent_t hp_event;

hipError_t (*kernel_func)(const void* func, dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, hipStream_t stream) = NULL;
hipError_t (*memcpy_func)(void* dst, const void* src, size_t count, enum hipMemcpyKind kind) = NULL;
hipError_t (*memcpy_async_func)(void* dst, const void* src, size_t count, enum hipMemcpyKind kind, hipStream_t stream) = NULL;
hipError_t (*malloc_func)(void** devPtr, size_t size) = NULL;
hipError_t (*free_func)(void* devPtr) = NULL;
hipError_t (*memset_func)(void* devPtr, int  value, size_t count) = NULL;
hipError_t (*memset_async_func)(void* devPtr, int  value, size_t count, hipStream_t stream) = NULL;

hipdnnStatus_t (*cudnn_conv_func)(hipdnnHandle_t handle, const void *alpha, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionFwdAlgo_t algo, void *workSpace, size_t workSpaceSizeInBytes, const void *beta, const hipdnnTensorDescriptor_t yDesc, void *y) = NULL;
hipdnnStatus_t (*cudnn_bnorm_func)(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, cudnnBatchNormOps_t bnOps, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *xData, const hipdnnTensorDescriptor_t zDesc,  const void *zData, const hipdnnTensorDescriptor_t yDesc, void *yData, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScaleData, const void *bnBiasData, double exponentialAverageFactor, void *resultRunningMeanData, void *resultRunningVarianceData, double epsilon, void *saveMean, void *saveInvVariance, const hipdnnActivationDescriptor_t activationDesc,  void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes) = NULL;
hipdnnStatus_t (*cudnn_bnorm_infer_func)(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnTensorDescriptor_t yDesc, void *y, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScale, const void *bnBias, const void *estimatedMean, const void *estimatedVariance, double epsilon) = NULL;
hipdnnStatus_t (*cudnn_rnn_func)(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes) = NULL;
hipdnnStatus_t (*cudnn_rnn_train_func)(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes) = NULL;
hipblasStatus_t (*cublas_sgemm_func)(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const float *alpha, const float *A, int lda, const float *B, int ldb, const float *beta, float *C, int ldc) = NULL;
hipblasStatus_t (*cublas_sgemm_strided_func)(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const float *alpha, const float *A, int lda, long long int strideA, const float *B, int ldb, long long int strideB, const float *beta, float *C, int ldc, long long int strideC, int batchCount) = NULL;

hipdnnStatus_t (*cudnn_bnorm_bw_func)(
	hipdnnHandle_t handle,
	hipdnnBatchNormMode_t mode,
	cudnnBatchNormOps_t bnOps,
	const void *alphaDataDiff,
    const void *betaDataDiff,
    const void *alphaParamDiff,
    const void *betaParamDiff,
    const hipdnnTensorDescriptor_t xDesc,
    const void *xData,
    const hipdnnTensorDescriptor_t yDesc,
    const void *yData,
    const hipdnnTensorDescriptor_t dyDesc,
    const void *dyData,
    const hipdnnTensorDescriptor_t dzDesc,
    void *dzData,
    const hipdnnTensorDescriptor_t dxDesc,
    void *dxData,
    const hipdnnTensorDescriptor_t dBnScaleBiasDesc,
    const void *bnScaleData,
    const void *bnBiasData,
    void *dBnScaleData,
    void *dBnBiasData,
    double epsilon,
    const void *savedMean,
    const void *savedInvVariance,
    const hipdnnActivationDescriptor_t activationDesc,
    void *workspace,
    size_t workSpaceSizeInBytes,
    void *reserveSpace,
    size_t reserveSpaceSizeInBytes
);

hipdnnStatus_t (*cudnn_conv_bw_data_func)(
	hipdnnHandle_t handle,
    const void *alpha,
    const hipdnnFilterDescriptor_t wDesc,
    const void *w,
    const hipdnnTensorDescriptor_t dyDesc,
    const void *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    hipdnnConvolutionBwdDataAlgo_t algo,
    void *workSpace,
    size_t workSpaceSizeInBytes,
    const void *beta,
    const hipdnnTensorDescriptor_t dxDesc,
    void *dx
);

hipdnnStatus_t (*cudnn_conv_bw_filter_func)(
	hipdnnHandle_t handle,
    const void *alpha,
    const hipdnnTensorDescriptor_t xDesc,
    const void *x,
    const hipdnnTensorDescriptor_t dyDesc,
    const void *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    hipdnnConvolutionBwdFilterAlgo_t algo,
    void *workSpace,
    size_t workSpaceSizeInBytes,
    const void *beta,
    const hipdnnFilterDescriptor_t dwDesc,
    void *dw
);

void print_kernel_invocation(int i, dim3 gridDim, dim3 blockDim) {

	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d], ", i);
	if (gridDim.y == 1 && gridDim.z == 1) {
  		DEBUG_PRINT("--gridDim=%d ", gridDim.x);
	} else if (gridDim.z == 1) {
		DEBUG_PRINT("--gridDim=[%d,%d] ", gridDim.x, gridDim.y);
	} else {
		DEBUG_PRINT("--gridDim=[%d,%d,%d] ", gridDim.x, gridDim.y, gridDim.z);
	}

	if (blockDim.y == 1 && blockDim.z == 1) {
		DEBUG_PRINT("--blockDim=%d ", blockDim.x);
	} else if (blockDim.z == 1) {
		DEBUG_PRINT("--blockDim=[%d,%d] ", blockDim.x, blockDim.y);
	} else {
		DEBUG_PRINT("--blockDim=[%d,%d,%d] ", blockDim.x, blockDim.y, blockDim.z);
	}
DEBUG_PRINT("\n");
}

void init() {

	pid_t pid = getpid();

	// map shared memory regions to talk with the scheduler

	// for requests
	std::string shmem_string_name = "client" + std::to_string(pid);
    const char* shmem_name = shmem_string_name.c_str();
	shared_memory_object shm (open_only, shmem_name, read_write);
    region = new mapped_region(shm, read_write);
	shmem = (int*)(region->get_address());

	// for streams
	std::string streams_shmem_string_name = "client_streams" + std::to_string(pid);
    const char* streams_shmem_name = streams_shmem_string_name.c_str();
	shared_memory_object streams_shm(open_only, streams_shmem_name, read_write);
    streams_region = new mapped_region(streams_shm, read_write);
	streams_shmem = (int*)(streams_region->get_address());

	// create streams
	int* lp = (int*)malloc(sizeof(int));
	int* hp = (int*)malloc(sizeof(int));

	CHECK_CUDA_ERROR(hipDeviceGetStreamPriorityRange(lp, hp));
	hipStreamCreateWithPriority(&lp_stream, hipStreamNonBlocking, 0);
	hipStreamCreateWithPriority(&hp_stream, hipStreamNonBlocking, *hp);

	printf("Highest stream priority is %d, lowest stream priority is %d\n", *hp, *lp);
	printf("LP stream: %d, hp stream: %d\n", lp_stream, hp_stream);

	CHECK_CUDA_ERROR(hipEventCreateWithFlags(&lp_event, hipEventDisableTiming));
	CHECK_CUDA_ERROR(hipEventCreateWithFlags(&hp_event, hipEventDisableTiming));

}

std::pair<hipStream_t, hipEvent_t> push_and_wait(int value, bool wait_for_stream) {

	*shmem = value;
	while (*shmem == value);
	if (wait_for_stream) {
		while (*streams_shmem == -1);
		hipStream_t sched_stream = (*streams_shmem == 0) ? lp_stream : hp_stream;
		hipEvent_t sched_event = (*streams_shmem == 0) ? lp_event : hp_event;
		*streams_shmem = -1;
		std::pair<hipStream_t, hipEvent_t> sched_pair(sched_stream, sched_event);
		return sched_pair;
	}
	else {
		std::pair<hipStream_t, hipEvent_t> sched_pair(lp_stream, lp_event);
		return sched_pair;
	}
}

void wait_events(hipStream_t stream) {
	CHECK_CUDA_ERROR(hipStreamWaitEvent(stream, lp_event, 0));
	CHECK_CUDA_ERROR(hipStreamWaitEvent(stream, hp_event, 0));
}

hipError_t hipMalloc(void** devPtr, size_t size) {

	if (shmem==NULL) {
		init();
	}

	push_and_wait(MALLOC_RECORD, false);

	int idx = get_idx();
	assert (idx >= 0);
	DEBUG_PRINT("[IDX %d] Caught hipMalloc! allocate region of %ld bytes\n", idx, size);

	hipError_t err = hipSuccess;

	if (malloc_func == NULL) {
		*(void **)(&malloc_func) = dlsym (RTLD_NEXT, "hipMalloc");
		assert (malloc_func != NULL);
	}

	//CHECK_CUDA_ERROR(hipDeviceSynchronize());
	err = (*malloc_func)(devPtr, size);
	CHECK_CUDA_ERROR(err);
	hipError_t err_all = hipDeviceSynchronize();
	CHECK_CUDA_ERROR(err_all);
	return err;

}


hipError_t hipFree(void* devPtr) {

	int idx = get_idx();
	assert (idx >= 0);
	DEBUG_PRINT("[IDX %d] Caught hipFree! free up address of %p\n", idx, devPtr);

	hipError_t err = hipSuccess;

	if (free_func == NULL) {
		*(void **)(&free_func) = dlsym (RTLD_NEXT, "hipFree");
		assert (free_func != NULL);
	}

	push_and_wait(FREE_RECORD, false);

	err = (*free_func)(devPtr);
	CHECK_CUDA_ERROR(err);

	return err;

}



hipError_t hipMemcpy(void* dst, const void* src, size_t count, enum hipMemcpyKind kind) {

	int idx = get_idx();
	assert (idx >= 0);
	DEBUG_PRINT("[IDX: %d], Caught hipMemcpy!\n", idx);

	hipError_t err = hipSuccess;

	if (memcpy_func == NULL) {
		*(void **)(&memcpy_func) = dlsym (RTLD_NEXT, "hipMemcpy");
		assert (memcpy_func != NULL);
	}

	err = (*memcpy_func)(dst, src, count, kind);
	CHECK_CUDA_ERROR(err);
	hipError_t err_all = hipDeviceSynchronize();
	CHECK_CUDA_ERROR(err_all);

	return err;

}


hipError_t hipMemcpyAsync(void* dst, const void* src, size_t count, enum hipMemcpyKind kind, hipStream_t stream) {

	int idx = get_idx();
	assert (idx >= 0);

	DEBUG_PRINT("[IDX: %d] Caught hipMemcpyAsync! src is %p, dst is %p, size is %d, stream is %d\n", idx, src, dst, count, stream);

	if (memcpy_async_func == NULL) {
		*(void **)(&memcpy_async_func) = dlsym (RTLD_NEXT, "hipMemcpyAsync");
		assert (memcpy_async_func != NULL);
	}

	hipError_t err = hipSuccess;
	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(MEMCPY_RECORD, true);

	wait_events(sched_pair.first);
	err = (*memcpy_async_func)(dst, src, count, kind, sched_pair.first); // TODO: not sure about which stream to use here
	CHECK_CUDA_ERROR(err);
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));
	return err;

}


hipError_t hipMemset(void* devPtr, int  value, size_t count ) {

	int idx = get_idx();
	assert (idx >= 0);

	DEBUG_PRINT("[IDX: %d] Caught hipMemset!\n", idx);
	hipError_t err = hipSuccess;

	if (memset_func == NULL) {
		*(void **)(&memset_func) = dlsym (RTLD_NEXT, "hipMemset");
		assert (memset_async_func != NULL);
	}

	err = (*memset_func)(devPtr, value, count);
	CHECK_CUDA_ERROR(err);

	return err;
}


hipError_t hipMemsetAsync ( void* devPtr, int  value, size_t count, hipStream_t stream) {

	int idx = get_idx();
	assert (idx >= 0);

	DEBUG_PRINT("[IDX: %d] Caught hipMemsetAsync!\n", idx);

	if (memset_async_func == NULL) {
		*(void **)(&memset_async_func) = dlsym (RTLD_NEXT, "hipMemsetAsync");
		assert (memset_async_func != NULL);
	}

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(MEMSET_RECORD, true);
	wait_events(sched_pair.first);

	hipError_t err = hipSuccess;
	err = (*memset_async_func)(devPtr, value, count, sched_pair.first);
	CHECK_CUDA_ERROR(err);
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));
	return err;

}


hipError_t hipLaunchKernel(reinterpret_cast<const void*>( const void* func), dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, hipStream_t stream ) {


	int idx = get_idx();
	assert (idx >= 0);

	// TODO: remove this
	// if (idx < 2)
	// 	block(idx,  mutexes, kqueues);

	//if (idx < 2)
	//	DEBUG_PRINT("------------------------- IDX %d, model name is %s\n", idx, model_names[idx]);

	//DEBUG_PRINT("[INTERCEPTER-CATCH-%d] Captured a hipLaunchKernel! function ptr is %p, stream is %d, gridDim is %d, blockDim is %d, sharedMem is %ld\n", idx, func, stream, gridDim, blockDim, sharedMem);
	//print_kernel_invocation(func_indexes[idx], gridDim, blockDim);

	if (kernel_func == NULL) {
		*(void **)(&kernel_func) = dlsym (RTLD_NEXT, "hipLaunchKernel");
		assert (kernel_func != NULL);
	}

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(KERNEL_RECORD, true);
	wait_events(sched_pair.first);

	hipError_t err = hipSuccess;
	kernel_record new_kernel_record;
	bool wait = false;

	DEBUG_PRINT("[INTERCEPTER] about to submit %p\n", func);

	err = (*kernel_func)(func, gridDim, blockDim, args, sharedMem, sched_pair.first);
	CHECK_CUDA_ERROR(err); // this checks kernel-launching errors
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));
	return err;
}