
#include "intercept_temp.h"

hipblasStatus_t hipblasSgemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const float *alpha, const float *A, int lda, const float *B, int ldb, const float *beta, float *C, int ldc) {

	int idx = get_idx();
	assert (idx >= 0);
	hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

	if (cublas_sgemm_func==NULL) {
		*(void **)(&cublas_sgemm_func) = dlsym(RTLD_NEXT, "hipblasSgemm");
		assert(cublas_sgemm_func != NULL);
	}

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(CUBLAS_SGEMM_RECORD, true);
	wait_events(sched_pair.first);
	hipblasSetStream(handle, sched_pair.first);

	status = (*cublas_sgemm_func)(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	assert (status == HIPBLAS_STATUS_SUCCESS);
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));
	DEBUG_PRINT("CUBLAS status is %d\n", status);
	return status;

}


hipblasStatus_t hipblasSgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const float *alpha, const float *A, int lda, long long int strideA, const float *B, int ldb, long long int strideB, const float *beta, float *C, int ldc, long long int strideC, int batchCount) {

	int idx = get_idx();
	assert (idx >= 0);
	hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

	if (cublas_sgemm_strided_func==NULL) {
		*(void **)(&cublas_sgemm_strided_func) = dlsym(RTLD_NEXT, "hipblasSgemmStridedBatched");
		assert(cublas_sgemm_strided_func != NULL);
	}

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(CUBLAS_SGEMM_STRIDED_RECORD, true);
	wait_events(sched_pair.first);
	hipblasSetStream(handle, sched_pair.first);

	status = (*cublas_sgemm_strided_func)(handle, transa, transb, m, n, k, alpha, A, lda, strideA, B, ldb, strideB, beta, C, ldc, strideC, batchCount);
	assert (status == HIPBLAS_STATUS_SUCCESS);
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));
	DEBUG_PRINT("CUBLAS status is %d\n", status);
	return status;
}

hipblasStatus_t hipblasDestroy(hipblasHandle_t handle) {

	DEBUG_PRINT("Caught a hipblasDestroy! Do nothing!\n");
	return HIPBLAS_STATUS_SUCCESS;
}
