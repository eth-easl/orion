#include "intercept_temp.h"

using namespace std;
using at::native::ReduceOp;
using at::_isnan;

template <typename acc_t>
struct MaxNanFunctor {
       	__device__ __forceinline__ acc_t operator()(acc_t a, acc_t b) const {
		return (at::_isnan(a) || a > b) ? a : b;
	}
};

template <typename acc_t>
struct MinNanFunctor {
	  __device__ __forceinline__ acc_t operator()(acc_t a, acc_t b) const {
		return (at::_isnan(a) || a < b) ? a : b;
	  }
};

template <typename T>
T* create_new_reduce_arg(void* args0) {

	T* reduce_arg = (T*)args0;
	T* new_reduce_arg = (T*)malloc(sizeof(T));
	char* dst0 = (char*)(reduce_arg->dst[0]);
	char* dst1 = (char*)(reduce_arg->dst[1]);

	*new_reduce_arg = T(
		reduce_arg->ops,
		reduce_arg->config,
		reduce_arg->input_calc,
		reduce_arg->output_calc,
		reduce_arg->src,
		dst0,
		dst1, //check this
		reduce_arg->acc_buf,
		reduce_arg->cta_buf,
		reduce_arg->semaphores,
		reduce_arg->ident,
		reduce_arg->noutputs,
		reduce_arg->base_idx
	);

	return new_reduce_arg;

}


queue<func_record> kqueue0;
queue<func_record> kqueue1;
pthread_mutex_t mutex0;
pthread_mutex_t mutex1;

vector<char*> fnames0;
vector<char*> fnames1;
volatile pid_t thread_ids[3]; // N threads + scheduler

queue<func_record>* kqueues[2] = {&kqueue0, &kqueue1};
pthread_mutex_t* mutexes[2] = {&mutex0, &mutex1};
vector<char*>* func_names[2] = {&fnames0, &fnames1};
char* model_names[2];

int func_indexes[2] = {0, 0};

hipStream_t client_streams[2];
bool streams_set[2] = {false, false};

hipError_t (*kernel_func)(const void* func, dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, hipStream_t stream) = NULL;
hipError_t (*memcpy_func)(void* dst, const void* src, size_t count, enum hipMemcpyKind kind) = NULL;
hipError_t (*memcpy_async_func)(void* dst, const void* src, size_t count, enum hipMemcpyKind kind, hipStream_t stream) = NULL;
hipError_t (*malloc_func)(void** devPtr, size_t size) = NULL;
hipError_t (*free_func)(void* devPtr) = NULL;
hipdnnStatus_t (*cudnn_conv_func)(hipdnnHandle_t handle, const void *alpha, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionFwdAlgo_t algo, void *workSpace, size_t workSpaceSizeInBytes, const void *beta, const hipdnnTensorDescriptor_t yDesc, void *y) = NULL;
hipdnnStatus_t (*cudnn_bnorm_func)(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, cudnnBatchNormOps_t bnOps, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *xData, const hipdnnTensorDescriptor_t zDesc,  const void *zData, const hipdnnTensorDescriptor_t yDesc, void *yData, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScaleData, const void *bnBiasData, double exponentialAverageFactor, void *resultRunningMeanData, void *resultRunningVarianceData, double epsilon, void *saveMean, void *saveInvVariance, const hipdnnActivationDescriptor_t activationDesc,  void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes) = NULL;
hipdnnStatus_t (*cudnn_bnorm_infer_func)(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnTensorDescriptor_t yDesc, void *y, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScale, const void *bnBias, const void *estimatedMean, const void *estimatedVariance, double epsilon) = NULL;
hipdnnStatus_t (*cudnn_rnn_func)(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes) = NULL;
hipdnnStatus_t (*cudnn_rnn_train_func)(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes) = NULL;
hipblasStatus_t (*cublas_sgemm_func)(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const float *alpha, const float *A, int lda, const float *B, int ldb, const float *beta, float *C, int ldc) = NULL;
hipblasStatus_t (*cublas_sgemm_strided_func)(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const float *alpha, const float *A, int lda, long long int strideA, const float *B, int ldb, long long int strideB, const float *beta, float *C, int ldc, long long int strideC, int batchCount) = NULL;


void print_kernel_invocation(int i, dim3 gridDim, dim3 blockDim) {

	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d], ", i);
	if (gridDim.y == 1 && gridDim.z == 1) {
  		DEBUG_PRINT("--gridDim=%d ", gridDim.x);
	} else if (gridDim.z == 1) {
		DEBUG_PRINT("--gridDim=[%d,%d] ", gridDim.x, gridDim.y);
	} else {
		DEBUG_PRINT("--gridDim=[%d,%d,%d] ", gridDim.x, gridDim.y, gridDim.z);
	}

	if (blockDim.y == 1 && blockDim.z == 1) {
		DEBUG_PRINT("--blockDim=%d ", blockDim.x);
	} else if (blockDim.z == 1) {
		DEBUG_PRINT("--blockDim=[%d,%d] ", blockDim.x, blockDim.y);
	} else {
		DEBUG_PRINT("--blockDim=[%d,%d,%d] ", blockDim.x, blockDim.y, blockDim.z);
	}
DEBUG_PRINT("\n");
}

hipError_t hipMalloc(void** devPtr, size_t size) {


	int idx = get_idx();
	assert (idx >= 0);
	DEBUG_PRINT("[IDX %d] Caught hipMalloc! allocate region of %ld bytes\n", idx, size);

	hipError_t err = hipSuccess;

	if (malloc_func == NULL) {
		*(void **)(&malloc_func) = dlsym (RTLD_NEXT, "hipMalloc");
		assert (malloc_func != NULL);
	}

	if (idx < 2) {

		//wait for all kernels or memory operations to finish
		block(idx,  mutexes, kqueues);

		malloc_record new_malloc_record = {devPtr, size};
		union func_data new_func_data;
		new_func_data.malrecord = new_malloc_record;
		func_record new_record = {MALLOC_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		// wait for mem to be allocated
		block(idx,  mutexes, kqueues);
		DEBUG_PRINT("[IDX %d] Exit malloc!\n", idx);
	}

	else {
		//CHECK_CUDA_ERROR(hipDeviceSynchronize());
		err = (*malloc_func)(devPtr, size);
		CHECK_CUDA_ERROR(err);
		hipError_t err_all = hipDeviceSynchronize();
		CHECK_CUDA_ERROR(err_all);
	}

	return err;

}


hipError_t hipMallocManaged(void** devPtr, size_t size, unsigned int flags) {

	//DEBUG_PRINT("Caught cudaMallocMANAGED! allocate region of %ld bytes\n", size);

	hipError_t (*function)(void** devPtr, size_t size, unsigned int flags);
	*(void **)(&function) = dlsym (RTLD_NEXT, "hipMallocManaged");

	hipError_t err = (*function)(devPtr, size, flags);
	CHECK_CUDA_ERROR(err);
	//DEBUG_PRINT("Memory allocated at address %p, size is %ld\n", *devPtr, size);
	return err;

}


hipError_t hipFree(void* devPtr) {

	int idx = get_idx();
	assert (idx >= 0);
	DEBUG_PRINT("[IDX %d] Caught hipFree! free up address of %p\n", idx, devPtr);

	hipError_t err = hipSuccess;

	if (free_func == NULL) {
		*(void **)(&free_func) = dlsym (RTLD_NEXT, "hipFree");
		assert (free_func != NULL);
	}

	if (idx < 2) {

		// wait for all kernels or memory operations to finish
		block(idx,  mutexes, kqueues);
		free_record new_free_record = {devPtr};

		union func_data new_func_data;
		new_func_data.frecord = new_free_record;
		func_record new_record = {FREE_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

	}
	else {
		err = (*free_func)(devPtr);
		CHECK_CUDA_ERROR(err);
	}

	return err;

}



hipError_t hipMemcpy(void* dst, const void* src, size_t count, enum hipMemcpyKind kind) {

	int idx = get_idx();
	assert (idx >= 0);
	DEBUG_PRINT("[IDX: %d], Caught hipMemcpy!\n", idx);

	hipError_t err = hipSuccess;

	if (memcpy_func == NULL) {
		*(void **)(&memcpy_func) = dlsym (RTLD_NEXT, "hipMemcpy");
		assert (memcpy_func != NULL);
	}

	if (idx < 2) {

		// wait for all kernels or memory operations to finish
		block(idx,  mutexes, kqueues);
		memcpy_record new_memcpy_record = {dst, src, count, kind, 0, false};

		union func_data new_func_data;
		new_func_data.mrecord = new_memcpy_record;
		func_record new_record = {MEMCPY_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		// wait for memcpy to finish
		block(idx,  mutexes, kqueues);
	}
	else {

		err = (*memcpy_func)(dst, src, count, kind);
		CHECK_CUDA_ERROR(err);
		hipError_t err_all = hipDeviceSynchronize();
		CHECK_CUDA_ERROR(err_all);

	}

	return err;

}


hipError_t hipMemcpyAsync(void* dst, const void* src, size_t count, enum hipMemcpyKind kind, hipStream_t stream) {

	int idx = get_idx();
	assert (idx >= 0);

	DEBUG_PRINT("[IDX: %d] Caught hipMemcpyAsync! src is %p, dst is %p, size is %d, stream is %d\n", idx, src, dst, count, stream);

	if (memcpy_async_func == NULL) {
		*(void **)(&memcpy_async_func) = dlsym (RTLD_NEXT, "hipMemcpyAsync");
		assert (memcpy_async_func != NULL);
	}

	hipError_t err = hipSuccess;

	if (idx < 2) {

		//wait for all kernels or memory operations to finish
		block(idx,  mutexes, kqueues);

		memcpy_record new_memcpy_record = {dst, src, count, kind, stream, true};

		union func_data new_func_data;
		new_func_data.mrecord = new_memcpy_record;
		func_record new_record = {MEMCPY_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		// although async, wait for debugging purposes
		block(idx,  mutexes, kqueues);
	}
	else {
		//CHECK_CUDA_ERROR(hipDeviceSynchronize());
		err = (*memcpy_async_func)(dst, src, count, kind, stream); // TODO: not sure about which stream to use here
		//err = (*function)(dst, src, count, kind);
		CHECK_CUDA_ERROR(err);
		//hipError_t err_all = hipDeviceSynchronize(); // although async, wait for debugging purposes
		//CHECK_CUDA_ERROR(err_all);
	}

	return err;

}


hipError_t hipMemset(void* devPtr, int  value, size_t count ) {

	printf("----------- Caught CUDA_MEMSET!!!!!!!!!!!!!\n");
	hipError_t (*function)(void* devPtr, int value, size_t count);
	*(void **)(&function) = dlsym (RTLD_NEXT, "hipMemset");

	hipError_t err = (*function)(devPtr, value, count);
	CHECK_CUDA_ERROR(err);
	return err;
}


hipError_t hipMemsetAsync ( void* devPtr, int  value, size_t count, hipStream_t stream) {

	printf("----------- Caught CUDA_MEMSET_ASYNC!!!!!!!!!!!!!\n");
	hipError_t (*function)(void* devPtr, int value, size_t count, hipStream_t stream);
	*(void **)(&function) = dlsym (RTLD_NEXT, "hipMemsetAsync");

	hipError_t err = (*function)(devPtr, value, count, stream);
	CHECK_CUDA_ERROR(err);
	return err;


}


hipError_t hipLaunchKernel(reinterpret_cast<const void*>( const void* func), dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, hipStream_t stream ) {


	int idx = get_idx();
	assert (idx >= 0);

	// TODO: remove this
	// if (idx < 2)
	// 	block(idx,  mutexes, kqueues);

	//if (idx < 2)
	//	DEBUG_PRINT("------------------------- IDX %d, model name is %s\n", idx, model_names[idx]);

	DEBUG_PRINT("[INTERCEPTER-CATCH-%d] Captured a hipLaunchKernel! function ptr is %p, stream is %d, gridDim is %d, blockDim is %d, sharedMem is %ld\n", idx, func, stream, gridDim, blockDim, sharedMem);
	//print_kernel_invocation(func_indexes[idx], gridDim, blockDim);

	if (kernel_func == NULL) {
		*(void **)(&kernel_func) = dlsym (RTLD_NEXT, "hipLaunchKernel");
		assert (kernel_func != NULL);
	}

	hipError_t err = hipSuccess;
	kernel_record new_kernel_record;
	bool wait = false;

	if (idx < 2) {

		pthread_mutex_lock(mutexes[idx]);

		// TODO: get kernel name correctly here
		char* kernel_name = func_names[idx]->at(func_indexes[idx]);
		DEBUG_PRINT("[INTERCEPTER] found a new kernel id %d, name is %s, func pointer is %p\n", func_indexes[idx], kernel_name, func);

		if (!strncmp(kernel_name, VECTORIZED_ELEMENTWISE_KERNEL, 41)) {

			// NOTE: WE EXPECT AN ADDITIONAL ARGUMENT WITH THE NUMBER OF INPUT/OUTPUT TENSORS
			// TODO: How to get this during runtime?
			void** new_args = (void**)malloc(4*sizeof(void*));
			// first arg: int

 			int* first_arg = (int*)malloc(sizeof(int));
			new_args[0] = first_arg;
			*first_arg = *((int*)(args[0]));

			new_args[1] = args[1];

			int data_size = *((int*)args[2]);
			int* data_size_ptr = (int*)malloc(sizeof(int));
			*data_size_ptr = data_size;
			Array<char*, 10>* data_ptr = (Array<char*, 10>*)malloc(sizeof(Array<char*, 10>));
			for (int i=0; i<data_size; i++) {
				data_ptr->data[i] = ((Array<char*, 10>*)args[3])->data[i];
			}

			new_args[2] = data_size_ptr;
			new_args[3] = data_ptr;


			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};

			//wait = true;
		}
		else if (!strncmp(kernel_name, CUB_DEVICE_REDUCE_SINGLE_TILE_KERNEL, 54)) {

			void** new_args = (void**)malloc(5*sizeof(void*));
			new_args[0] = args[0];
			new_args[1] = args[1];

			new_args[2] = (int*)malloc(sizeof(int));
			*((int*)new_args[2]) = *((int*)(args[2]));

			new_args[3] = args[3];

			new_args[4] = (int*)malloc(sizeof(int));
			*((int*)new_args[4]) = *((int*)(args[4]));

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			//wait = true;

		}
		else if (!strncmp(kernel_name, CUB_DEVICE_COMPACT_INIT_KERNEL, 49)) {

			void** new_args = (void**)malloc(3*sizeof(void*));

			new_args[0] = args[0];

			new_args[1] = (int*)malloc(sizeof(int));
			*((int*)new_args[1]) = *((int*)(args[1]));

			new_args[2] = args[2];

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			//wait = true;
		}
		else if (!strncmp(kernel_name, CUB_DEVICE_SELECT_SWEEP_KERNEL, 49)) {

			void** new_args = (void**)malloc(9*sizeof(void*));
			for (int i=0; i<7; i++)
				new_args[i] = args[i];

			new_args[7] = (int*)malloc(sizeof(int));
			*((int*)new_args[7]) = *((int*)(args[7]));

			new_args[8] = (int*)malloc(sizeof(int));
			*((int*)new_args[8]) = *((int*)(args[8]));

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			//wait = true;

		}
		else if (!strncmp(kernel_name, INDEX_ELEMENTWISE_KERNEL, 41)) {

			void** new_args = (void**)malloc(2*sizeof(void*));

			new_args[0] = (int*)malloc(sizeof(int));
			*((int*)new_args[0]) = *((int*)(args[0]));

			new_args[1] = args[1];

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			//wait = true; // leave this for now
		}
		else if (!strncmp(kernel_name, UNROLLED_ELEMENTWISE_KERNEL, 44)) {

			// NOTE: WE EXPECT AN ADDITIONAL ARGUMENT WITH THE NUMBER OF INPUT/OUTPUT TENSORS
			// TODO: How to get this during runtime?

			void** new_args = (void**)malloc(8*sizeof(void*));
			new_args[0] = (int*)malloc(sizeof(int));
			*((int*)new_args[0]) = *((int*)(args[0]));

			new_args[1] = args[1];
			new_args[4] = args[4];
			new_args[5] = args[5];
			new_args[6] = args[6];
			new_args[7] = args[7];

			int data_size = *((int*)args[2]);

			int* data_size_ptr = (int*)malloc(sizeof(int));
			*data_size_ptr = data_size;
			Array<char*, 10>* data_ptr = (Array<char*, 10>*)malloc(sizeof(Array<char*, 10>));
			for (int i=0; i<data_size; i++) {
				data_ptr->data[i] = ((Array<char*, 10>*)args[3])->data[i];
			}

			new_args[2] = data_size_ptr;
			new_args[3] = data_ptr;

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			// TODO: why BERT has problem here?
			//wait = true;
		}
		else if (!strncmp(kernel_name, REDUCE_KERNEL, 44)) {

			void** new_args = (void**)malloc(sizeof(void*));
			if (!strcmp(model_names[idx], RESNET50) && (func_indexes[idx] == 225 or func_indexes[idx] == 172)) {
				using arg_type = at::native::ReduceOp<float, at::native::MeanOps<float, float>, unsigned int, float, 4>;
				arg_type* new_reduce_arg = create_new_reduce_arg<arg_type>(args[0]);
				new_args[0] = new_reduce_arg;
			}
			else if (!strcmp(model_names[idx], MOBILENET) && (func_indexes[idx] == 149 or func_indexes[idx] == 201)) {

				using arg_type = at::native::ReduceOp<float, at::native::MeanOps<float, float>, unsigned int, float, 4>;
				arg_type* new_reduce_arg = create_new_reduce_arg<arg_type>(args[0]);
				new_args[0] = new_reduce_arg;
			}
			else if (!strcmp(model_names[idx], GNMT) && func_indexes[idx] == 35) {
				using arg_type = at::native::ReduceOp<float, at::native::NormTwoOps<float, float>, unsigned int, float, 4>;
				arg_type* new_reduce_arg = create_new_reduce_arg<arg_type>(args[0]);
				new_args[0] = new_reduce_arg;

			}
			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};

			//wait = true;
		}
		else if (!strncmp(kernel_name, MAX_POOL_FORWARD_NCHW, 61)) {

			void** new_args = (void**)malloc(17*sizeof(void*));

			new_args[0]  = (int*)malloc(sizeof(int));
			*((int*)new_args[0]) = *((int*)(args[0]));
			for (int i=2; i<15; i++) {
				new_args[i] = (int*)malloc(sizeof(int));
				*((int*)new_args[i]) = *((int*)(args[i]));
			}
			new_args[1] = args[1];
			new_args[15] = args[15];
			new_args[16] = args[16];

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};

			// TODO: check why invalid memory accesses here (for both reads and writes)
			//wait = true;
		}
		else if (!strncmp(kernel_name, ELEMENTWISE_KERNEL_WITH_INDEX, 57)) {

			void** new_args = (void**)malloc(3*sizeof(void*));
			new_args[0]  = (int*)malloc(sizeof(int));
			*(((int*)new_args[0])) = *((int*)(args[0]));

			new_args[1] = args[1];
			new_args[2] = args[2];

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			// used in bert, only once so just wait
			//wait = true;
		}
		else if (!strncmp(kernel_name, INDEX_SELECT_LARGE_INDEX, 61)) {

			void** new_args = (void**)malloc(8*sizeof(void*));
			new_args[0] = args[0];
			new_args[1] = args[1];
			new_args[2] = args[2];

			for (int i=3; i<5; i++) {
				new_args[i] = (int*)malloc(sizeof(int));
				*(((int*)new_args[i])) = *((int*)(args[i]));
			}

			for (int i=5; i<7; i++) {
				new_args[i] = (unsigned int*)malloc(sizeof(unsigned int));
				*(((unsigned int*)new_args[i])) = *((unsigned int*)(args[i]));
			}

			new_args[7] = (int64_t*)malloc(sizeof(int64_t));
			*(((int64_t*)new_args[7])) = *((int64_t*)(args[7]));

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			// invalid memory acces - why?
			//wait = true;
		}
		else if (!strncmp(kernel_name, ELEMENTWISE_KERNEL, 35)) {

			void** new_args = (void**)malloc(8*sizeof(void*));

			new_args[0] = (int*)malloc(sizeof(int));
			*(((int*)new_args[0])) = *((int*)(args[0]));

			new_args[1] = args[1];
			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			// TODO: VERY IMPORTANT - invalid memory access - why?
			//wait = true;
		}
		else if (!strncmp(kernel_name, SOFTMAX_WARP_FORWARD, 48)) {

			void** new_args = (void**)malloc(8*sizeof(void*));
			new_args[0] = args[0];
			new_args[1] = args[1];
			new_args[5] = args[5];

			for (int i=2; i<5; i++) {
				new_args[i] = (int*)malloc(sizeof(int));
				*(((int*)new_args[i])) = *((int*)(args[i]));
			}

			new_args[6] = (int*)malloc(sizeof(int));
			*(((int*)new_args[6])) = *((int*)(args[6]));

			new_args[7] = (bool*)malloc(sizeof(bool));
			*(((bool*)new_args[7])) = *((bool*)(args[7]));

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			// TODO: FIXME!
			//wait = true;
		}
		else if (!strncmp(kernel_name, VECTORIZED_LAYER_NORM_KERNEL, 68)) {

			void** new_args = (void**)malloc(8*sizeof(void*));

			new_args[0] = (int*)malloc(sizeof(int));
			*(((int*)new_args[0])) = *((int*)(args[0]));

			new_args[1] = (float*)malloc(sizeof(float));
			*(((float*)new_args[1])) = *((int*)(args[1]));

			for (int i=2; i<8; i++)
				new_args[i] = args[i];

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			// TODO: FIXME!!!!!!
			//wait = true;
		}
		else if (!strncmp(kernel_name, TRIU_TRIL_KERNEL, 33)) {

			void** new_args = (void**)malloc(4*sizeof(void*));
			new_args[0] = args[0];
			new_args[1] = args[1];

			for (int i=2; i<4; i++) {
				new_args[i] = (int64_t*)malloc(sizeof(int64_t));
				*(((int64_t*)new_args[i])) = *((int64_t*)(args[i]));
			}
			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			//wait = true;
		}
		else if (!strncmp(kernel_name, CAT_ARRAY_BATCHED_COPY, 59)) {

			void** new_args = (void**)malloc(5*sizeof(void*));
			for (int i=0; i<3; i++)
				new_args[i] = args[i];

			new_args[3] = (int*)malloc(sizeof(int));
			*(((int*)new_args[3])) = *((int*)(args[3]));

			new_args[4] = (unsigned int*)malloc(sizeof(unsigned int));
			*(((unsigned int*)new_args[4])) = *((unsigned int*)(args[4]));

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			// TODO: FIXME
			//wait = true;
		}
		else if (!strncmp(kernel_name, UPSAMPLE_BILINEAR2D_OUT_FRAME, 69)) {

			void** new_args = (void**)malloc(6*sizeof(void*));

			new_args[0] = (int*)malloc(sizeof(int));
			*(((int*)new_args[0])) = *((int*)(args[0]));

			new_args[1] = (float*)malloc(sizeof(float));
			*(((float*)new_args[1])) = *((float*)(args[1]));

			new_args[2] = (float*)malloc(sizeof(float));
			*(((float*)new_args[2])) = *((float*)(args[2]));

			new_args[3] = (bool*)malloc(sizeof(bool));
			*(((bool*)new_args[3])) = *((bool*)(args[3]));

			new_args[4] = args[4];
			new_args[5] = args[5];

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};

			// TODO: FIXME
			//wait = true;
		}
		else if (!strncmp(kernel_name, UPSAMPLE_NEAREST2D_NHWC_OUT_FRAME, 73)) {

			void** new_args = (void**)malloc(10*sizeof(void*));

			new_args[0] = args[0];
			new_args[1] = args[1];

			for (int i=2; i<7; i++){
				new_args[i] = (size_t*)malloc(sizeof(size_t));
				*(((size_t*)new_args[i])) = *((size_t*)(args[i]));
			}

			new_args[9] = (size_t*)malloc(sizeof(size_t));
			*(((size_t*)new_args[9])) = *((size_t*)(args[9]));

			for (int i=7; i<9; i++){
				new_args[i] = (float*)malloc(sizeof(float));
				*(((float*)new_args[i])) = *((float*)(args[i]));
			}

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			// TODO: FIXME
			//wait = true;
		}
		else if (!strncmp(kernel_name, CUB_DEVICE_REDUCE_KERNEL, 44)) {

			void** new_args = (void**)malloc(5*sizeof(void*));
			new_args[0] = args[0];
			new_args[1] = args[1];
			new_args[3] = args[3];
			new_args[4] = args[4];

			new_args[2] = (int*)malloc(sizeof(int));
			*(((int*)new_args[2])) = *((int*)(args[2]));

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			//wait = true;
		}
		else if (!strncmp(kernel_name, CUB_DEVICE_SCAN_INIT_KERNEL, 46)) {

			void** new_args = (void**)malloc(2*sizeof(void*));
			new_args[0] = args[0];

			new_args[1] = (size_t*)malloc(sizeof(size_t));
			*(((size_t*)new_args[1])) = *((size_t*)(args[1]));

			new_kernel_record = {func, gridDim, blockDim, new_args, sharedMem, stream, false, 0};
			//wait = true;
		}
		else if (!strncmp(kernel_name, CUB_DEVICE_SCAN_KERNEL, 42)) {


			new_kernel_record = {func, gridDim, blockDim, args, sharedMem, stream, false, 0};
			//wait = true;

		}
		else {

			new_kernel_record = {func, gridDim, blockDim, args, sharedMem, stream, false, 0};
			wait = true;
		}

		//new_kernel_record = {func, gridDim, blockDim, args, sharedMem, stream, false, 0};
		union func_data new_func_data;
		new_func_data.krecord = new_kernel_record;
		func_record new_record = {KERNEL_RECORD, new_func_data};

		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;

		//if (wait)
		block(idx,  mutexes, kqueues);

	}
	else {
		DEBUG_PRINT("[INTERCEPTER] about to submit %p\n", func);

		err = (*kernel_func)(func, gridDim, blockDim, args, sharedMem, stream);
		DEBUG_PRINT("*************** [INTERCEPTER] AFTER SUBMITTING %p *************\n", func);
		CHECK_CUDA_ERROR(err); // this checks kernel-launching errors

		// hipError_t err_all = hipDeviceSynchronize(); // for debugging
		// CHECK_CUDA_ERROR(err_all); // this checks (or should check) runtime-specific errors

		// hipError_t err2 = hipGetLastError();
		// CHECK_CUDA_ERROR(err2);



	}
	return err;
}


// CUDNN ....


// CUBLAS ....
