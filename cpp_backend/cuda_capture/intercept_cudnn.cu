
#include "intercept_temp.h"

void getDescriptor(const hipdnnTensorDescriptor_t desc) {

	int ndims = 10;
	hipdnnDataType_t* dtype = (hipdnnDataType_t*)malloc(sizeof(hipdnnDataType_t));
	int* nbdims = (int*)malloc(sizeof(int));
	int dimA[10] = {0};
	int strideA[10] = {0};

	hipdnnStatus_t status = hipdnnGetTensorNdDescriptor(desc, ndims, dtype, nbdims, dimA, strideA);
	printf("%d\n", *dtype);

	assert (status==HIPDNN_STATUS_SUCCESS);

}

hipdnnStatus_t hipdnnConvolutionForward(hipdnnHandle_t handle, const void *alpha, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionFwdAlgo_t algo, void *workSpace, size_t workSpaceSizeInBytes, const void *beta, const hipdnnTensorDescriptor_t yDesc, void *y) {


	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	DEBUG_PRINT("[INTERCEPTER-CATCH-%d]-[%d] Caught hipdnnConvolutionForward, CUDNN handle is %p\n", idx, func_indexes[idx], handle, idx);

	// if (idx < 2)
	// 	block(idx,  mutexes, kqueues);

	cudnnConvolutionForward_record new_conv_record = {
		handle,
		alpha,
		xDesc,
		x,
		wDesc,
		w,
		convDesc,
		algo,
		workSpace,
		workSpaceSizeInBytes,
		beta,
		yDesc,
		y
	};
	union func_data new_func_data;
	new_func_data.cudnnConvRecord = new_conv_record;
	func_record new_record = {CUDNN_CONV_RECORD, new_func_data};

	// push or run
	if (idx < 2) {
		 pthread_mutex_lock(mutexes[idx]);
		 kqueues[idx]->push(new_record);
		 pthread_mutex_unlock(mutexes[idx]);

		 func_indexes[idx] += 1;
		 block(idx, mutexes, kqueues);
	}
	else {

		if (cudnn_conv_func==NULL) {
			*(void **)(&cudnn_conv_func) = dlsym(RTLD_NEXT, "hipdnnConvolutionForward");
			assert(cudnn_conv_func != NULL);
		}

		status = (*cudnn_conv_func)(handle, alpha, xDesc, x, wDesc, w, convDesc, algo, workSpace, workSpaceSizeInBytes, beta, yDesc, y);
		if (status != HIPDNN_STATUS_SUCCESS)
			printf("status is %d\n", status);
		assert (status == HIPDNN_STATUS_SUCCESS);

		DEBUG_PRINT("CONV submitted!!\n");
	}

	return status;

}

hipdnnStatus_t cudnnBatchNormalizationForwardTrainingEx(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, cudnnBatchNormOps_t bnOps, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *xData, const hipdnnTensorDescriptor_t zDesc,  const void *zData, const hipdnnTensorDescriptor_t yDesc, void *yData, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScaleData, const void *bnBiasData, double exponentialAverageFactor, void *resultRunningMeanData, void *resultRunningVarianceData, double epsilon, void *saveMean, void *saveInvVariance, const hipdnnActivationDescriptor_t activationDesc,  void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes) {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	DEBUG_PRINT("[INTERCEPTER-CATCH-%d]-[%d] Caught cudnnBatchNormalizationForwardTrainingEx, handle is %p\n", idx, func_indexes[idx], handle);

	// if (idx < 2)
	// 	block(idx,  mutexes, kqueues);

	// create record
	cudnnBatchNormalizationForwardTrainingEx_record new_bn_record = {
		handle,
		mode,
		bnOps,
		alpha,
		beta,
		xDesc,
		xData,
		zDesc,
		zData,
		yDesc,
		yData,
		bnScaleBiasMeanVarDesc,
		bnScaleData,
		bnBiasData,
		exponentialAverageFactor,
		resultRunningMeanData,
		resultRunningVarianceData,
		epsilon,
		saveMean,
		saveInvVariance,
		activationDesc,
		workspace,
		workSpaceSizeInBytes,
		reserveSpace,
		reserveSpaceSizeInBytes

	};
	union func_data new_func_data;
	new_func_data.cudnnBNormRecord = new_bn_record;
	func_record new_record = {CUDNN_BNORM_RECORD, new_func_data};

	// push or run

	if (idx < 2) {

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;
		block(idx,  mutexes, kqueues);
	}
	else {

		if (cudnn_bnorm_func==NULL) {
			*(void **)(&cudnn_bnorm_func) = dlsym(RTLD_NEXT, "cudnnBatchNormalizationForwardTrainingEx");
			assert(cudnn_bnorm_func != NULL);
		}
		//printf("run func %p\n", cudnn_bnorm_function);
		status = (*cudnn_bnorm_func)(handle, mode, bnOps, alpha, beta, xDesc, xData, zDesc, zData, yDesc, yData, bnScaleBiasMeanVarDesc, bnScaleData, bnBiasData, exponentialAverageFactor, resultRunningMeanData, resultRunningVarianceData, epsilon, saveMean, saveInvVariance, activationDesc, workspace, workSpaceSizeInBytes, reserveSpace, reserveSpaceSizeInBytes);
		// if (status != HIPDNN_STATUS_SUCCESS)
		// 	printf("status is %d\n", status);
		// assert (status == HIPDNN_STATUS_SUCCESS);

		// DEBUG_PRINT("BNORM submitted!!\n");

	}

	return status;
}


hipdnnStatus_t hipdnnBatchNormalizationForwardInference(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnTensorDescriptor_t yDesc, void *y, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScale, const void *bnBias, const void *estimatedMean, const void *estimatedVariance, double epsilon)

{

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnBatchNormalizationForwardInference, handle is %p, index is %d\n", func_indexes[idx], handle, idx);

	// if (idx < 2)
	// 	block(idx,  mutexes, kqueues);


	// create record
	cudnnBatchNormalizationForwardInference_record bn_record = {
		handle,
		mode,
		alpha,
		beta,
		xDesc,
		x,
		yDesc,
		y,
		bnScaleBiasMeanVarDesc,
		bnScale,
		bnBias,
		estimatedMean,
		estimatedVariance,
		epsilon
	};

	union func_data new_func_data;
	new_func_data.cudnnBNormInfRecord = bn_record;
	func_record new_record = {CUDNN_BNORM_INF_RECORD, new_func_data};

	if (idx < 2) {

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;
		block(idx,  mutexes, kqueues);

	}
	else {

		if (cudnn_bnorm_infer_func==NULL) {
			*(void **)(&cudnn_bnorm_infer_func) = dlsym(RTLD_NEXT, "hipdnnBatchNormalizationForwardInference");
			assert(cudnn_bnorm_infer_func != NULL);
		}

		status = (*cudnn_bnorm_infer_func)(handle, mode, alpha, beta, xDesc, x, xDesc, y, bnScaleBiasMeanVarDesc, bnScale, bnBias, estimatedMean, estimatedVariance, epsilon);
		assert (status == HIPDNN_STATUS_SUCCESS);

	}

	return status;
}


hipdnnStatus_t hipdnnRNNForwardInference(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes)  {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;


	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnRNNForwardInference, handle is %p, index is %d\n", func_indexes[idx], handle, idx);
	printf("------------------------------------------------- IDX [%d], CX IS %p, CY IS %p\n", idx, cx, cy);

	if (idx < 2) {

		hipdnnTensorDescriptor_t* xDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
	        //hipdnnStatus_t s = hipdnnCreateTensorDescriptor(xDesc_new);

		*xDesc_new = *xDesc;
		//printf("%p, %p, %p, %p\n", xDesc, *xDesc, xDesc_new, *(xDesc_new));
		//memcpy(xDesc_new, xDesc, sizeof(hipdnnTensorDescriptor_t));

		hipdnnTensorDescriptor_t* yDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
		*yDesc_new = *yDesc;


		cudnnRNNForwardInf_record rnn_record = {
			handle,
			rnnDesc,
			seqLength,
			xDesc_new,
			x,
			hxDesc,
			hx,
			cxDesc,
			cx,
			wDesc,
			w,
			yDesc_new,
			y,
			hyDesc,
			hy,
			cyDesc,
			cy,
			workspace,
			workSpaceSizeInBytes
		};

		union func_data new_func_data;
		new_func_data.cudnnRnnInfRecord = rnn_record;
		func_record new_record = {CUDNN_RNN_INF_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;
		block(idx,  mutexes, kqueues);
	}
	else {

		if (cudnn_rnn_func==NULL) {
			*(void **)(&cudnn_rnn_func) = dlsym(RTLD_NEXT, "hipdnnRNNForwardInference");
			assert(cudnn_rnn_func != NULL);
		}

		status = (*cudnn_rnn_func)(handle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSpaceSizeInBytes);

		printf("------------------------- cudnn status is %d\n", status);
		// TODO: not sure why this complains here in just one call!
		assert (status == HIPDNN_STATUS_SUCCESS);

		// hipError_t err_all = hipDeviceSynchronize(); // for debugging
		// CHECK_CUDA_ERROR(err_all);
	}

	return status;

}


hipdnnStatus_t hipdnnRNNForwardTraining(
	hipdnnHandle_t handle,
	const hipdnnRNNDescriptor_t rnnDesc,
	const int seqLength,
	const hipdnnTensorDescriptor_t *xDesc,
	const void *x,
    const hipdnnTensorDescriptor_t hxDesc,
    const void *hx,
    const hipdnnTensorDescriptor_t cxDesc,
    const void *cx,
    const hipdnnFilterDescriptor_t wDesc,
    const void *w,
    const hipdnnTensorDescriptor_t *yDesc,
    void *y,
    const hipdnnTensorDescriptor_t hyDesc,
    void *hy,
    const hipdnnTensorDescriptor_t cyDesc,
    void *cy,
    void *workspace,
    size_t workSpaceSizeInBytes,
    void *reserveSpace,
    size_t reserveSpaceSizeInBytes
) {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	DEBUG_PRINT("[INTERCEPTER-CATCH]-[%d] Caught hipdnnRNNForwardTraining, handle is %p, index is %d\n", func_indexes[idx], handle, idx);

	if (idx < 2) {

		hipdnnTensorDescriptor_t* xDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
	        //hipdnnStatus_t s = hipdnnCreateTensorDescriptor(xDesc_new);

		*xDesc_new = *xDesc;
		printf("%p, %p, %p, %p\n", xDesc, *xDesc, xDesc_new, *(xDesc_new));
		//memcpy(xDesc_new, xDesc, sizeof(hipdnnTensorDescriptor_t));

		hipdnnTensorDescriptor_t* yDesc_new = (hipdnnTensorDescriptor_t*)malloc(sizeof(hipdnnTensorDescriptor_t));
		*yDesc_new = *yDesc;

		cudnnRNNForwardTraining_record rnn_record = {
			handle,
			rnnDesc,
			seqLength,
			xDesc,
			x,
			hxDesc,
			hx,
			cxDesc,
			cx,
			wDesc,
			w,
			yDesc,
			y,
			hyDesc,
			hy,
			cyDesc,
			cy,
			workspace,
			workSpaceSizeInBytes,
			reserveSpace,
			reserveSpaceSizeInBytes
		};

		union func_data new_func_data;
		new_func_data.cudnnRnnTrainRecord = rnn_record;
		func_record new_record = {CUDNN_RNN_TRAIN_RECORD, new_func_data};

		pthread_mutex_lock(mutexes[idx]);
		kqueues[idx]->push(new_record);
		pthread_mutex_unlock(mutexes[idx]);

		func_indexes[idx] += 1;
		block(idx,  mutexes, kqueues);
	}
	else {

		if (cudnn_rnn_train_func==NULL) {
			*(void **)(&cudnn_rnn_train_func) = dlsym(RTLD_NEXT, "hipdnnRNNForwardTraining");
			assert(cudnn_rnn_train_func != NULL);
		}

		status = (*cudnn_rnn_train_func)(handle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSpaceSizeInBytes, reserveSpace, reserveSpaceSizeInBytes);

		// hipError_t err_all = hipDeviceSynchronize(); // for debugging
		// CHECK_CUDA_ERROR(err_all);
	}

	return status;
}


hipdnnStatus_t hipdnnDestroyRNNDescriptor(hipdnnRNNDescriptor_t rnnDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyRNNDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyTensorDescriptor(hipdnnTensorDescriptor_t tensorDesc) {

	// mock cudnn destroy TensorDescriptor
	//DEBUG_PRINT("Caught a hipdnnDestroyTensorDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}


hipdnnStatus_t hipdnnDestroyFilterDescriptor(hipdnnFilterDescriptor_t filterDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyFilterDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;

}


hipdnnStatus_t hipdnnDestroyConvolutionDescriptor(hipdnnConvolutionDescriptor_t convDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyConvolutionDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyDropoutDescriptor(hipdnnDropoutDescriptor_t dropoutDesc) {
	//DEBUG_PRINT("Caught a hipdnnDestroyDropoutDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;

}


hipdnnStatus_t hipdnnDestroy(hipdnnHandle_t handle) {

	printf("Caught a hipdnnDestroy, Do nothing!\n ");
	return HIPDNN_STATUS_SUCCESS;
}