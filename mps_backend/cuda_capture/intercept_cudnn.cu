
#include "intercept_temp.h"

hipdnnStatus_t hipdnnConvolutionForward(hipdnnHandle_t handle, const void *alpha, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionFwdAlgo_t algo, void *workSpace, size_t workSpaceSizeInBytes, const void *beta, const hipdnnTensorDescriptor_t yDesc, void *y) {


	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	if (cudnn_conv_func==NULL) {
		*(void **)(&cudnn_conv_func) = dlsym(RTLD_NEXT, "hipdnnConvolutionForward");
		assert(cudnn_conv_func != NULL);
	}

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(CUDNN_CONV_RECORD, true);
	wait_events(sched_pair.first);
	hipdnnSetStream(handle, sched_pair.first);

	status = (*cudnn_conv_func)(handle, alpha, xDesc, x, wDesc, w, convDesc, algo, workSpace, workSpaceSizeInBytes, beta, yDesc, y);
	if (status != HIPDNN_STATUS_SUCCESS)
		printf("status is %d\n", status);
	assert (status == HIPDNN_STATUS_SUCCESS);
	DEBUG_PRINT("CONV submitted!!\n");
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));

	return status;

}

hipdnnStatus_t cudnnBatchNormalizationForwardTrainingEx(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, cudnnBatchNormOps_t bnOps, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *xData, const hipdnnTensorDescriptor_t zDesc,  const void *zData, const hipdnnTensorDescriptor_t yDesc, void *yData, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScaleData, const void *bnBiasData, double exponentialAverageFactor, void *resultRunningMeanData, void *resultRunningVarianceData, double epsilon, void *saveMean, void *saveInvVariance, const hipdnnActivationDescriptor_t activationDesc,  void *workspace, size_t workSpaceSizeInBytes, void *reserveSpace, size_t reserveSpaceSizeInBytes) {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	if (cudnn_bnorm_func==NULL) {
		*(void **)(&cudnn_bnorm_func) = dlsym(RTLD_NEXT, "cudnnBatchNormalizationForwardTrainingEx");
		assert(cudnn_bnorm_func != NULL);
	}

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(CUDNN_BNORM_RECORD, true);
	wait_events(sched_pair.first);
	hipdnnSetStream(handle, sched_pair.first);

	//printf("run func %p\n", cudnn_bnorm_function);
	status = (*cudnn_bnorm_func)(handle, mode, bnOps, alpha, beta, xDesc, xData, zDesc, zData, yDesc, yData, bnScaleBiasMeanVarDesc, bnScaleData, bnBiasData, exponentialAverageFactor, resultRunningMeanData, resultRunningVarianceData, epsilon, saveMean, saveInvVariance, activationDesc, workspace, workSpaceSizeInBytes, reserveSpace, reserveSpaceSizeInBytes);
	assert (status == HIPDNN_STATUS_SUCCESS);
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));

	return status;
}


hipdnnStatus_t hipdnnBatchNormalizationForwardInference(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void *alpha, const void *beta, const hipdnnTensorDescriptor_t xDesc, const void *x, const hipdnnTensorDescriptor_t yDesc, void *y, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void *bnScale, const void *bnBias, const void *estimatedMean, const void *estimatedVariance, double epsilon)

{

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;
	// if (idx < 2)
	// 	block(idx,  mutexes, kqueues);

	if (cudnn_bnorm_infer_func==NULL) {
		*(void **)(&cudnn_bnorm_infer_func) = dlsym(RTLD_NEXT, "hipdnnBatchNormalizationForwardInference");
		assert(cudnn_bnorm_infer_func != NULL);
	}

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(CUDNN_BNORM_INF_RECORD, true);
	wait_events(sched_pair.first);
	hipdnnSetStream(handle, sched_pair.first);

	status = (*cudnn_bnorm_infer_func)(handle, mode, alpha, beta, xDesc, x, xDesc, y, bnScaleBiasMeanVarDesc, bnScale, bnBias, estimatedMean, estimatedVariance, epsilon);
	assert (status == HIPDNN_STATUS_SUCCESS);
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));

	return status;
}


hipdnnStatus_t hipdnnRNNForwardInference(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t *xDesc, const void *x, const hipdnnTensorDescriptor_t hxDesc, const void *hx, const hipdnnTensorDescriptor_t cxDesc, const void *cx, const hipdnnFilterDescriptor_t wDesc, const void *w, const hipdnnTensorDescriptor_t *yDesc, void *y, const hipdnnTensorDescriptor_t hyDesc, void *hy, const hipdnnTensorDescriptor_t cyDesc, void *cy, void *workspace, size_t workSpaceSizeInBytes)  {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	if (cudnn_rnn_func==NULL) {
		*(void **)(&cudnn_rnn_func) = dlsym(RTLD_NEXT, "hipdnnRNNForwardInference");
		assert(cudnn_rnn_func != NULL);
	}

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(CUDNN_RNN_INF_RECORD, true);
	wait_events(sched_pair.first);
	hipdnnSetStream(handle, sched_pair.first);

	status = (*cudnn_rnn_func)(handle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSpaceSizeInBytes);
	// TODO: not sure why this complains here in just one call!
	assert (status == HIPDNN_STATUS_SUCCESS);
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));

	return status;

}


hipdnnStatus_t hipdnnRNNForwardTraining(
	hipdnnHandle_t handle,
	const hipdnnRNNDescriptor_t rnnDesc,
	const int seqLength,
	const hipdnnTensorDescriptor_t *xDesc,
	const void *x,
    const hipdnnTensorDescriptor_t hxDesc,
    const void *hx,
    const hipdnnTensorDescriptor_t cxDesc,
    const void *cx,
    const hipdnnFilterDescriptor_t wDesc,
    const void *w,
    const hipdnnTensorDescriptor_t *yDesc,
    void *y,
    const hipdnnTensorDescriptor_t hyDesc,
    void *hy,
    const hipdnnTensorDescriptor_t cyDesc,
    void *cy,
    void *workspace,
    size_t workSpaceSizeInBytes,
    void *reserveSpace,
    size_t reserveSpaceSizeInBytes
) {

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	if (cudnn_rnn_train_func==NULL) {
		*(void **)(&cudnn_rnn_train_func) = dlsym(RTLD_NEXT, "hipdnnRNNForwardTraining");
		assert(cudnn_rnn_train_func != NULL);
	}

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(CUDNN_RNN_TRAIN_RECORD, true);
	wait_events(sched_pair.first);
	hipdnnSetStream(handle, sched_pair.first);

	status = (*cudnn_rnn_train_func)(handle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSpaceSizeInBytes, reserveSpace, reserveSpaceSizeInBytes);
	assert (status == HIPDNN_STATUS_SUCCESS);
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));

	return status;
}


// backward

hipdnnStatus_t cudnnBatchNormalizationBackwardEx (
    hipdnnHandle_t handle,
    hipdnnBatchNormMode_t mode,
    cudnnBatchNormOps_t bnOps,
    const void *alphaDataDiff,
    const void *betaDataDiff,
    const void *alphaParamDiff,
    const void *betaParamDiff,
    const hipdnnTensorDescriptor_t xDesc,
    const void *xData,
    const hipdnnTensorDescriptor_t yDesc,
    const void *yData,
    const hipdnnTensorDescriptor_t dyDesc,
    const void *dyData,
    const hipdnnTensorDescriptor_t dzDesc,
    void *dzData,
    const hipdnnTensorDescriptor_t dxDesc,
    void *dxData,
    const hipdnnTensorDescriptor_t dBnScaleBiasDesc,
    const void *bnScaleData,
    const void *bnBiasData,
    void *dBnScaleData,
    void *dBnBiasData,
    double epsilon,
    const void *savedMean,
    const void *savedInvVariance,
    const hipdnnActivationDescriptor_t activationDesc,
    void *workspace,
    size_t workSpaceSizeInBytes,
    void *reserveSpace,
    size_t reserveSpaceSizeInBytes
) {

	int idx = get_idx();
	assert (idx >= 0);

	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	if (cudnn_bnorm_bw_func==NULL) {
		*(void **)(&cudnn_bnorm_bw_func) = dlsym(RTLD_NEXT, "cudnnBatchNormalizationBackwardEx");
		assert(cudnn_bnorm_bw_func != NULL);
	}

	status = (*cudnn_bnorm_bw_func)(
			handle,
			mode,
			bnOps,
			alphaDataDiff,
			betaDataDiff,
			alphaParamDiff,
			betaParamDiff,
			xDesc,
			xData,
			yDesc,
			yData,
			dyDesc,
			dyData,
			dzDesc,
			dzData,
			dxDesc,
			dxData,
			dBnScaleBiasDesc,
			bnScaleData,
			bnBiasData,
			dBnScaleData,
			dBnBiasData,
			epsilon,
			savedMean,
			savedInvVariance,
			activationDesc,
			workspace,
			workSpaceSizeInBytes,
			reserveSpace,
			reserveSpaceSizeInBytes
	);

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(CUDNN_BNORM_BACKWARD_RECORD, true);
	wait_events(sched_pair.first);
	hipdnnSetStream(handle, sched_pair.first);

	assert (status == HIPDNN_STATUS_SUCCESS);
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));

	return status;
}

hipdnnStatus_t hipdnnConvolutionBackwardData(
    hipdnnHandle_t handle,
    const void *alpha,
    const hipdnnFilterDescriptor_t wDesc,
    const void *w,
    const hipdnnTensorDescriptor_t dyDesc,
    const void *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    hipdnnConvolutionBwdDataAlgo_t algo,
    void *workSpace,
    size_t workSpaceSizeInBytes,
    const void *beta,
    const hipdnnTensorDescriptor_t dxDesc,
    void *dx
)
{

	int idx = get_idx();
	assert (idx >= 0);
	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	if (cudnn_conv_bw_data_func==NULL) {
		*(void **)(&cudnn_conv_bw_data_func) = dlsym(RTLD_NEXT, "hipdnnConvolutionBackwardData");
		assert(cudnn_conv_bw_data_func != NULL);
	}
	status = (*cudnn_conv_bw_data_func)(
			handle,
			alpha,
			wDesc,
			w,
			dyDesc,
			dy,
			convDesc,
			algo,
			workSpace,
			workSpaceSizeInBytes,
			beta,
			dxDesc,
			dx
	);

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(CUDNN_CONV_DATA_RECORD, true);
	wait_events(sched_pair.first);
	hipdnnSetStream(handle, sched_pair.first);

	assert (status == HIPDNN_STATUS_SUCCESS);
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));
	return status;
}

hipdnnStatus_t hipdnnConvolutionBackwardFilter(
    hipdnnHandle_t handle,
    const void *alpha,
    const hipdnnTensorDescriptor_t xDesc,
    const void *x,
    const hipdnnTensorDescriptor_t dyDesc,
    const void *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    hipdnnConvolutionBwdFilterAlgo_t algo,
    void *workSpace,
    size_t workSpaceSizeInBytes,
    const void *beta,
    const hipdnnFilterDescriptor_t dwDesc,
    void *dw
) {

	int idx = get_idx();
	assert (idx >= 0);

	hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

	if (cudnn_conv_bw_filter_func==NULL) {
		*(void **)(&cudnn_conv_bw_filter_func) = dlsym(RTLD_NEXT, "hipdnnConvolutionBackwardFilter");
		assert(cudnn_conv_bw_filter_func != NULL);
	}

	status = (*cudnn_conv_bw_filter_func)(
			handle,
			alpha,
			xDesc,
			x,
			dyDesc,
			dy,
			convDesc,
			algo,
			workSpace,
			workSpaceSizeInBytes,
			beta,
			dwDesc,
			dw
	);

	std::pair<hipStream_t, hipEvent_t> sched_pair = push_and_wait(CUDNN_CONV_FILTER_RECORD, true);
	wait_events(sched_pair.first);
	hipdnnSetStream(handle, sched_pair.first);

	assert (status == HIPDNN_STATUS_SUCCESS);
	CHECK_CUDA_ERROR(hipEventRecord(sched_pair.second, sched_pair.first));

	return status;
}


hipdnnStatus_t hipdnnDestroyActivationDescriptor(hipdnnActivationDescriptor_t activationDesc) {
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyRNNDescriptor(hipdnnRNNDescriptor_t rnnDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyRNNDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyTensorDescriptor(hipdnnTensorDescriptor_t tensorDesc) {

	// mock cudnn destroy TensorDescriptor
	//DEBUG_PRINT("Caught a hipdnnDestroyTensorDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}


hipdnnStatus_t hipdnnDestroyFilterDescriptor(hipdnnFilterDescriptor_t filterDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyFilterDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;

}


hipdnnStatus_t hipdnnDestroyConvolutionDescriptor(hipdnnConvolutionDescriptor_t convDesc) {

	//DEBUG_PRINT("Caught a hipdnnDestroyConvolutionDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;
}

hipdnnStatus_t hipdnnDestroyDropoutDescriptor(hipdnnDropoutDescriptor_t dropoutDesc) {
	//DEBUG_PRINT("Caught a hipdnnDestroyDropoutDescriptor! Do nothing!\n");
	return HIPDNN_STATUS_SUCCESS;

}


hipdnnStatus_t hipdnnDestroy(hipdnnHandle_t handle) {

	printf("Caught a hipdnnDestroy, Do nothing!\n ");
	return HIPDNN_STATUS_SUCCESS;
}